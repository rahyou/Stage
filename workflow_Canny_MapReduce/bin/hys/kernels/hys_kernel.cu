
#include <hip/hip_runtime.h>
#ifdef __cplusplus
extern "C" {
#endif

__global__ void hys_kernel( int* data,
                               int* out,
                                       int rows,
                                       int cols)

{	float lowThresh = 60;
	float highThresh = 170;

	int g_row = threadIdx.y + (blockIdx.y * blockDim.y);
	int g_col = threadIdx.x + (blockIdx.x * blockDim.x);
int pos = g_col * cols + g_row;



   const int EDGE = 16777215;

   int magnitude = data[pos];
    
    if (magnitude >= highThresh)
        out[pos] = EDGE;
    else if (magnitude <= lowThresh)
        out[pos] =  EDGE;
    else
    {

       float med = (highThresh + lowThresh)/2;
        
        if (magnitude >= med)
            out[pos] = EDGE;
        else
             out[pos] = 0;
    }
}
#ifdef __cplusplus
}
#endif
