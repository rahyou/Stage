
#include <hip/hip_runtime.h>
#ifdef __cplusplus
extern "C" {
#endif
#define L_SIZE 16

__constant__ float gaus[3][3] =  { {0, 1, 0},
                                {1, -3, 1},
                                {0, 1, 0} };

__global__ void gauss_kernel( int* data,
                               int* out,
                                       int cols,int rows)
{
int g_row = blockIdx.x * blockDim.x + threadIdx.x;
int g_col = blockIdx.y * blockDim.y + threadIdx.y;
int pos = g_col * cols + g_row;

int l_row = threadIdx.x + 1;
int l_col = threadIdx.y + 1;
    
int sum=0;
  
  __shared__ int l_data[L_SIZE+2][L_SIZE+2];

    // copy to local
    l_data[l_row][l_col] = data[pos];

    // top most row
    if (l_row == 1)
    {
        l_data[0][l_col] = data[pos-cols];
        // top left
        if (l_col == 1)
            l_data[0][0] = data[pos-cols-1];

        // top right
        else if (l_col == L_SIZE)
            l_data[0][L_SIZE+1] = data[pos-cols+1];
    }
    // bottom most row
    else if (l_row == L_SIZE)
    {
        l_data[L_SIZE+1][l_col] = data[pos+cols];
        // bottom left
        if (l_col == 1)
            l_data[L_SIZE+1][0] = data[pos+cols-1];

        // bottom right
        else if (l_col == L_SIZE)
            l_data[L_SIZE+1][L_SIZE+1] = data[pos+cols+1];
    }

    if (l_col == 1)
        l_data[l_row][0] = data[pos-1];
    else if (l_col == L_SIZE)
        l_data[l_row][L_SIZE+1] = data[pos+1];

 

    for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
            sum += gaus[i][j] * l_data[i+l_row-1][j+l_col-1];


out[pos] = max(0,sum); ;

    return;
}

#ifdef __cplusplus
}
#endif
