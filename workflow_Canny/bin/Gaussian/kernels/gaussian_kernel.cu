#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

__constant float gaus[3][3] = { {0, 1, 0},
                                {1, -3, 1},
                                {0, 1, 0} };
#define L_SIZE 16

// Gaussian Kernel
// data: image input data with each pixel taking up 1 byte (8Bit 1Channel)
// out: image output data (8B1C)
__global__ void gauss_kernel( int* B,
                               int* out,
                                       int rows,
                                       int width)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= numberOfPixels){
		//printf("%d\n",index);
		return;
	}

	int mask[] = { 1, 2, 1, 2, 4, 2, 1, 2, 1 };
	int s = mask[0] + mask[1] + mask[2] + mask[3] + mask[4] + mask[5] + mask[6] + mask[7] + mask[8];

	if (index < width){ 
		if (index == 0){
			s = mask[4] + mask[1] + mask[2] + mask[5];
			out[index] = (int)((B[index] * mask[4] + B[index + width] * mask[1] + B[index + width + 1] * mask[2] + B[index + 1] * mask[5]) / s);
			
			return;
		}

		if (index == width - 1){
			s = mask[4] + mask[0] + mask[1] + mask[3];
			out[index] = (B[index] * mask[4] + B[index + width - 1] * mask[0] + B[index + width] * mask[1] + B[index - 1] * mask[3]);
		
			return;
		}
	
		s = mask[4] + mask[1] + mask[2] + mask[5] + mask[0] + mask[3];
		out[index] = (int)((B[index] * mask[4] + B[index + width] * mask[1] + B[index + width + 1] * mask[2] + B[index + 1] * mask[5] + B[index + width - 1] * mask[0] + B[index - 1] * mask[3]) / s);
		
		return;
	}
	if (index >= numberOfPixels - width){ 

		if (index == numberOfPixels - width){ 
			s = mask[4] + mask[5] + mask[7] + mask[8];
			out[index] = (int)((B[index] * mask[4] + B[index + 1] * mask[5] + B[index - width] * mask[7] + B[index - width + 1] * mask[8]) / s);
		
			return;
		}

		if (index == numberOfPixels - 1){ 
			s = mask[4] + mask[3] + mask[6] + mask[7];
			out[index] = (int)((B[index] * mask[4] + B[index - 1] * mask[3] + B[index - width - 1] * mask[6] + B[index - width] * mask[7]) / s);
			
			return;
		}

		s = mask[4] + mask[3] + mask[5] + mask[6] + mask[7] + mask[8];
		out[index] = (int)((B[index] * mask[4] + B[index - 1] * mask[3] + B[index - width - 1] * mask[6] + B[index - width] * mask[7] + B[index + 1] * mask[5] + B[index - width] * mask[8]) / s);
			return;
	}
	if (index % width == 0){ 
		s = mask[4] + mask[1] + mask[2] + mask[5] + mask[8] + mask[7];
		out[index] = (int)((B[index] * mask[4] + B[index + width] * mask[1] + B[index + width + 1] * mask[2] + B[index + 1] * mask[5] + B[index - width + 1] * mask[8] + B[index - width]) / s);
		
		return;
	}
#ifdef __cplusplus
}
#endif
