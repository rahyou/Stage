
#include <hip/hip_runtime.h>
#ifdef __cplusplus
extern "C" {
#endif


__global__ void vec_a(float *A, float* B,float* C,
				int size)
{
  int index = blockIdx.x*blockDim.x + threadIdx.x;

  if(index<size)
    C[index] = A[index] + B[index];
}

#ifdef __cplusplus
}
#endif
