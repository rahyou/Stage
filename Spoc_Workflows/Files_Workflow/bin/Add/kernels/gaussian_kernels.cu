#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif
__constant float gaus[3][3] = { {0.0625, 0.125, 0.0625},
                                {0.1250, 0.250, 0.1250},
                                {0.0625, 0.125, 0.0625} };

#define L_SIZE 16

// Gaussian Kernel
// data: image input data with each pixel taking up 1 byte (8Bit 1Channel)
// out: image output data (8B1C)
__global__ void gauss_kernel( int* data,
                               int* out,
                                       int rows,int cols)
{
int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index >= numberOfPixels){
		//printf("%d\n",index);
		return;
	}

	int mask[] = { 1, 2, 1, 2, 4, 2, 1, 2, 1 };
	int s = mask[0] + mask[1] + mask[2] + mask[3] + mask[4] + mask[5] + mask[6] + mask[7] + mask[8];

	if (index < width){ // dolny rzad pikseli
		if (index == 0){ //lewy dolny rog
			s = mask[4] + mask[1] + mask[2] + mask[5];
			B_new[index] = (int)((B[index] * mask[4] + B[index + width] * mask[1] + B[index + width + 1] * mask[2] + B[index + 1] * mask[5]) / s);
			G_new[index] = (int)((G[index] * mask[4] + G[index + width] * mask[1] + G[index + width + 1] * mask[2] + G[index + 1] * mask[5]) / s);
			R_new[index] = (int)((R[index] * mask[4] + R[index + width] * mask[1] + R[index + width + 1] * mask[2] + R[index + 1] * mask[5]) / s);
			return;
		}

		if (index == width - 1){//prawy dolny rog
			s = mask[4] + mask[0] + mask[1] + mask[3];
			B_new[index] = (B[index] * mask[4] + B[index + width - 1] * mask[0] + B[index + width] * mask[1] + B[index - 1] * mask[3]);
			G_new[index] = (G[index] * mask[4] + G[index + width - 1] * mask[0] + G[index + width] * mask[1] + G[index - 1] * mask[3]);
			R_new[index] = (R[index] * mask[4] + R[index + width - 1] * mask[0] + R[index + width] * mask[1] + R[index - 1] * mask[3]);
			return;
		}
		//reszta pikseli w dolnym rzedzie
		s = mask[4] + mask[1] + mask[2] + mask[5] + mask[0] + mask[3];
		B_new[index] = (int)((B[index] * mask[4] + B[index + width] * mask[1] + B[index + width + 1] * mask[2] + B[index + 1] * mask[5] + B[index + width - 1] * mask[0] + B[index - 1] * mask[3]) / s);
		R_new[index] = (int)((R[index] * mask[4] + R[index + width] * mask[1] + R[index + width + 1] * mask[2] + R[index + 1] * mask[5] + R[index + width - 1] * mask[0] + R[index - 1] * mask[3]) / s);
		G_new[index] = (int)((G[index] * mask[4] + G[index + width] * mask[1] + G[index + width + 1] * mask[2] + G[index + 1] * mask[5] + G[index + width - 1] * mask[0] + G[index - 1] * mask[3]) / s);

		return;
	}
	if (index >= numberOfPixels - width){ //gorny rzad pikseli

		if (index == numberOfPixels - width){ //lewy gorny rog
			s = mask[4] + mask[5] + mask[7] + mask[8];
			B_new[index] = (int)((B[index] * mask[4] + B[index + 1] * mask[5] + B[index - width] * mask[7] + B[index - width + 1] * mask[8]) / s);
			G_new[index] = (int)((G[index] * mask[4] + G[index + 1] * mask[5] + G[index - width] * mask[7] + G[index - width + 1] * mask[8]) / s);
			R_new[index] = (int)((R[index] * mask[4] + R[index + 1] * mask[5] + R[index - width] * mask[7] + R[index - width + 1] * mask[8]) / s);
			return;
		}

		if (index == numberOfPixels - 1){ //prawy gorny rog
			s = mask[4] + mask[3] + mask[6] + mask[7];
			B_new[index] = (int)((B[index] * mask[4] + B[index - 1] * mask[3] + B[index - width - 1] * mask[6] + B[index - width] * mask[7]) / s);
			G_new[index] = (int)((G[index] * mask[4] + G[index - 1] * mask[3] + G[index - width - 1] * mask[6] + G[index - width] * mask[7]) / s);
			R_new[index] = (int)((R[index] * mask[4] + R[index - 1] * mask[3] + R[index - width - 1] * mask[6] + R[index - width] * mask[7]) / s);
			return;
		}

		s = mask[4] + mask[3] + mask[5] + mask[6] + mask[7] + mask[8];
		B_new[index] = (int)((B[index] * mask[4] + B[index - 1] * mask[3] + B[index - width - 1] * mask[6] + B[index - width] * mask[7] + B[index + 1] * mask[5] + B[index - width] * mask[8]) / s);
		R_new[index] = (int)((R[index] * mask[4] + R[index - 1] * mask[3] + R[index - width - 1] * mask[6] + R[index - width] * mask[7] + R[index + 1] * mask[5] + R[index - width] * mask[8]) / s);
		G_new[index] = (int)((G[index] * mask[4] + G[index - 1] * mask[3] + G[index - width - 1] * mask[6] + G[index - width] * mask[7] + G[index + 1] * mask[5] + G[index - width] * mask[8]) / s);
		return;
	}
	if (index % width == 0){ //lewa sciana
		s = mask[4] + mask[1] + mask[2] + mask[5] + mask[8] + mask[7];
		B_new[index] = (int)((B[index] * mask[4] + B[index + width] * mask[1] + B[index + width + 1] * mask[2] + B[index + 1] * mask[5] + B[index - width + 1] * mask[8] + B[index - width]) / s);
		G_new[index] = (int)((G[index] * mask[4] + G[index + width] * mask[1] + G[index + width + 1] * mask[2] + G[index + 1] * mask[5] + G[index - width + 1] * mask[8] + G[index - width]) / s);
		R_new[index] = (int)((R[index] * mask[4] + R[index + width] * mask[1] + R[index + width + 1] * mask[2] + R[index + 1] * mask[5] + R[index - width + 1] * mask[8] + R[index - width]) / s);
		return;
	}
#ifdef __cplusplus
}
#endif
