
#include <hip/hip_runtime.h>
#ifdef __cplusplus
extern "C" {
#endif

__global__ void vec_add(float *A, float *B, float* C,
				int size)
{
  int index = blockIdx.x*blockDim.x + threadIdx.x;

  if(index<size)
    C[index] = A[index] + B[index];
}




__global__ void vec_add_double(double *A, double *B, double* C,
				int size)
{
  int index = blockIdx.x*blockDim.x + threadIdx.x;

  if(index<size)
    C[index] = A[index] + B[index];
}

#ifdef __cplusplus
}
#endif
